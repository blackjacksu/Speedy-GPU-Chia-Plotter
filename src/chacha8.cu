#include "hip/hip_runtime.h"
// CUDA libraries.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Include associated header file.
#include "../include/chacha8.cuh"

#define MAX_ARRAY_SIZE 256

// There is a multiply relation between the size of c (output) and n_blocks
// n_blocks = 1, sizeof(c) = uint8_t * 64 * 1
// n_blocks = 2, sizeof(c) = uint8_t * 64 * 2
// n_blocks = 3, sizeof(c) = uint8_t * 64 * 3
#define SIZE_OF_OUTPUT_PER_BLOCK 64

#define U32TO32_LITTLE(v) (v)
#define U8TO32_LITTLE(p) (*(const uint32_t *)(p))
#define U32TO8_LITTLE(p, v) (((uint32_t *)(p))[0] = U32TO32_LITTLE(v))
#define ROTL32(v, n) (((v) << (n)) | ((v) >> (32 - (n))))

#define ROTATE(v, c) (ROTL32(v, c))
#define XOR(v, w) ((v) ^ (w))
#define PLUS(v, w) ((v) + (w))
#define PLUSONE(v) (PLUS((v), 1))

#define QUARTERROUND(a, b, c, d) \
    a = PLUS(a, b);              \
    d = ROTATE(XOR(d, a), 16);   \
    c = PLUS(c, d);              \
    b = ROTATE(XOR(b, c), 12);   \
    a = PLUS(a, b);              \
    d = ROTATE(XOR(d, a), 8);    \
    c = PLUS(c, d);              \
    b = ROTATE(XOR(b, c), 7)

// This is the GPU device code
__global__ void Kernel_Print(int * block_dim, int * thread_id, int * grid_dim)
{
    printf("blockDim x:%d, y:%d, z:%d\n", blockDim.x, blockDim.y, blockDim.z);
    printf("threadIdx x:%d, y:%d, z:%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("gridDim x:%d, y:%d, z:%d\n", gridDim.x, gridDim.y, gridDim.z);

    block_dim[0] = blockDim.x;
    block_dim[1] = blockDim.y;
    block_dim[2] = blockDim.z;

    thread_id[0] = threadIdx.x;
    thread_id[1] = threadIdx.y;
    thread_id[2] = threadIdx.z;

    grid_dim[0] = gridDim.x;
    grid_dim[1] = gridDim.y;
    grid_dim[2] = gridDim.z;
}


// This is the GPU device code
__global__ void chacha8_get_keystream_cuda( struct chacha8_ctx *x, uint64_t *pos, uint64_t *n_blocks, uint8_t *_c, uint8_t *c_start, int array_size)
{
    int idx = threadIdx.x;
    printf("[chacha8_get_keystream_cuda] i = %d, array size:%d\n", idx, array_size);

    if (idx >= array_size)
    {
        // Out of bound
        return;
    }

    uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
    uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
    int i;
    uint8_t *c;
    c = &_c[c_start[idx]];

    j0 = x->input[0];
    j1 = x->input[1];
    j2 = x->input[2];
    j3 = x->input[3];
    j4 = x->input[4];
    j5 = x->input[5];
    j6 = x->input[6];
    j7 = x->input[7];
    j8 = x->input[8];
    j9 = x->input[9];
    j10 = x->input[10];
    j11 = x->input[11];
    j12 = pos[idx];
    j13 = pos[idx] >> 32;
    j14 = x->input[14];
    j15 = x->input[15];

    printf("x is clear j0=%d\n, j0");
    printf("x is clear j1=%d\n, j1");

    while (n_blocks[idx]--) {
        printf("n block is clear, n_blocks[idx]: %d\n", n_blocks[idx]);
        x0 = j0;
        x1 = j1;
        x2 = j2;
        x3 = j3;
        x4 = j4;
        x5 = j5;
        x6 = j6;
        x7 = j7;
        x8 = j8;
        x9 = j9;
        x10 = j10;
        x11 = j11;
        x12 = j12;
        x13 = j13;
        x14 = j14;
        x15 = j15;
        for (i = 8; i > 0; i -= 2) {
            QUARTERROUND(x0, x4, x8, x12);
            QUARTERROUND(x1, x5, x9, x13);
            QUARTERROUND(x2, x6, x10, x14);
            QUARTERROUND(x3, x7, x11, x15);
            QUARTERROUND(x0, x5, x10, x15);
            QUARTERROUND(x1, x6, x11, x12);
            QUARTERROUND(x2, x7, x8, x13);
            QUARTERROUND(x3, x4, x9, x14);
        }
        x0 = PLUS(x0, j0);
        x1 = PLUS(x1, j1);
        x2 = PLUS(x2, j2);
        x3 = PLUS(x3, j3);
        x4 = PLUS(x4, j4);
        x5 = PLUS(x5, j5);
        x6 = PLUS(x6, j6);
        x7 = PLUS(x7, j7);
        x8 = PLUS(x8, j8);
        x9 = PLUS(x9, j9);
        x10 = PLUS(x10, j10);
        x11 = PLUS(x11, j11);
        x12 = PLUS(x12, j12);
        x13 = PLUS(x13, j13);
        x14 = PLUS(x14, j14);
        x15 = PLUS(x15, j15);

        j12 = PLUSONE(j12);
        if (!j12) {
            j13 = PLUSONE(j13);
            /* stopping at 2^70 bytes per nonce is user's responsibility */
        }


        U32TO8_LITTLE(c + 0, x0); // c[0] = x0
        U32TO8_LITTLE(c + 4, x1); 
        U32TO8_LITTLE(c + 8, x2);
        U32TO8_LITTLE(c + 12, x3);
        U32TO8_LITTLE(c + 16, x4);
        U32TO8_LITTLE(c + 20, x5);
        U32TO8_LITTLE(c + 24, x6);
        U32TO8_LITTLE(c + 28, x7);
        U32TO8_LITTLE(c + 32, x8);
        U32TO8_LITTLE(c + 36, x9);
        U32TO8_LITTLE(c + 40, x10);
        U32TO8_LITTLE(c + 44, x11);
        U32TO8_LITTLE(c + 48, x12);
        U32TO8_LITTLE(c + 52, x13);
        U32TO8_LITTLE(c + 56, x14);
        U32TO8_LITTLE(c + 60, x15);

        c += 64;


        // printf("c is clear");
    }

}



void get_chacha8_key(struct chacha8_ctx *h_x, uint64_t *h_pos, uint64_t *h_n_blocks, uint8_t *h_c, uint64_t *h_c_start, uint64_t h_c_size, int h_array_size)
{
    // 
    // std::cout << "Size of uint64_t:" << sizeof(uint64_t) << std::endl;
    // std::cout << "Size of * uint64_t:" << sizeof(_pos) << std::endl;
    // std::cout << "Size of uint32_t:" << sizeof(uint32_t) << std::endl;
    // std::cout << "Size of * uint32_t:" << sizeof(_n_blocks) << std::endl;
    // std::cout << "Size of struct chacha8_ctx:" << sizeof(struct chacha8_ctx) << std::endl;
    std::cout << "h_x->input[5]: " << h_x->input[5] << std::endl;
    std::cout << "h_x->input[6]: " << h_x->input[6] << std::endl;
    std::cout << "h_x->input[7]: " << h_x->input[7] << std::endl;
    std::cout << "_pos[0]: " << h_pos[0] << std::endl;
    std::cout << "_pos[1]: " << h_pos[1] << std::endl;
    std::cout << "h_c_start[1]: " << h_c_start[1] << std::endl;
    std::cout << "h_c_start[2]: " << h_c_start[2] << std::endl;
    std::cout << "h_c_size: " << h_c_size << std::endl;
    std::cout << "h_array_size: " << h_array_size << std::endl;

    if (h_array_size > MAX_ARRAY_SIZE)
    {
        std::cout << "Array size out of bound" << std::endl;
        return;
    }

    struct chacha8_ctx *d_x;
    uint64_t *d_pos;
    uint64_t *d_n_blocks;
    uint8_t *d_c;
    uint8_t *d_c_start;
    int block_per_grid = 1;
    int thread_per_block = h_array_size;

    // Has to handle error if memory allocation failed
    hipError_t error;

    std::cout << "Array size: " << h_array_size << std::endl;
    // Allocate space for device
    error = hipMalloc((void**) &d_pos, h_array_size * sizeof(uint64_t));
    if (error)
    {
        std::cout << "hipMalloc fail at pos error: " << error << std::endl; 
        return;
    }

    error = hipMalloc((void**) &d_n_blocks, h_array_size * sizeof(uint64_t));
    if (error)
    {
        std::cout << "hipMalloc fail at n_blocks error: " << error << std::endl; 
        return;
    }

    error = hipMalloc((void**) &d_x, h_array_size * sizeof(struct chacha8_ctx));
    if (error)
    {
        std::cout << "hipMalloc fail at x error: " << error << std::endl; 
        return;
    }
    
    error = hipMalloc((void**) &d_c, h_c_size * sizeof(uint64_t));
    if (error)
    {
        std::cout << "hipMalloc fail at c error: " << error << std::endl; 
        return;
    }

    error = hipMalloc((void**) &d_c_start, h_array_size * sizeof(uint64_t));
    if (error)
    {
        std::cout << "hipMalloc fail at c_start error: " << error << std::endl; 
        return;
    }

    // Copy content from host to device
    error = hipMemcpy(d_pos, h_pos, h_array_size * sizeof(uint64_t), hipMemcpyHostToDevice);
    if (error)
    {
        std::cout << "[H->D]hipMemcpy fail at pos error: " << error << std::endl; 
        return;
    }

    error = hipMemcpy(d_n_blocks, h_n_blocks, h_array_size * sizeof(uint64_t), hipMemcpyHostToDevice);
    if (error)
    {
        std::cout << "[H->D]hipMemcpy fail at n_blocks error: " << error << std::endl; 
        return;
    }

    error = hipMemcpy(d_x, h_x, sizeof(struct chacha8_ctx), hipMemcpyHostToDevice);
    if (error)
    {
        std::cout << "[H->D]hipMemcpy fail at x error: " << error << std::endl; 
        return;
    }
    
    error = hipMemcpy(d_c_start, h_c_start, h_array_size * sizeof(uint64_t), hipMemcpyHostToDevice);
    if (error)
    {
        std::cout << "[H->D]hipMemcpy fail at x error: " << error << std::endl; 
        return;
    }

    chacha8_get_keystream_cuda<<<block_per_grid, thread_per_block>>>(d_x, d_pos, d_n_blocks, d_c, d_c_start, h_array_size);

    // std::cout << "Malloc and Memcpy done" << std::endl;
    // // std::cout << "x: " << x[0].input[0] << x[0].input[1] << std::endl;
    // std::cout << "n_blocks: " << n_blocks[0] << std::endl;
    // std::cout << "pos: " << pos[0] << std::endl;


    // // Calculate blocksize and gridsize.
    // // dim3 blockSize(512, 1, 1);
    // // dim3 gridSize(512 / array_size + 1, 1);



    // // Copy result to output
    error = hipMemcpy(h_c, d_c, h_c_size * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (error)
    {
        std::cout << "[H<-D]hipMemcpy fail at host c error: " << error << std::endl; 
        return;
    }

    int * block_dim;
    int * thread_id;
    int * grid_dim;

    hipMalloc((void**) &block_dim, 3 * sizeof(int));
    hipMalloc((void**) &thread_id, 3 * sizeof(int));
    hipMalloc((void**) &grid_dim, 3 * sizeof(int));

    int bd[3];
    int ti[3];
    int gd[3];


    Kernel_Print<<<block_per_grid, thread_per_block>>>(block_dim, thread_id, grid_dim);

    hipMemcpy(bd, block_dim, 3 * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "bd:" << bd[0] << bd[1] << bd[2] << std::endl;

    // // free memory
    // error = hipFree(d_x);
    // if (error)
    // {
    //     std::cout << "hipFree fail at d_x error: " << error << std::endl; 
    //     return;
    // }
    // error = hipFree(d_pos);
    // if (error)
    // {
    //     std::cout << "hipFree fail at d_pos error: " << error << std::endl; 
    //     return;
    // }
    // error = hipFree(d_n_blocks);
    // if (error)
    // {
    //     std::cout << "hipFree fail at d_n_blocks error: " << error << std::endl; 
    //     return;
    // }
}

// This is host code
void chacha8_keysetup(struct chacha8_ctx *x, const uint8_t *k, uint32_t kbits, const uint8_t *iv)
{
    const char *constants;
    static const char sigma[17] = "expand 32-byte k";
    static const char tau[17] = "expand 16-byte k";

    x->input[4] = U8TO32_LITTLE(k + 0);
    x->input[5] = U8TO32_LITTLE(k + 4);
    x->input[6] = U8TO32_LITTLE(k + 8);
    x->input[7] = U8TO32_LITTLE(k + 12);
    if (kbits == 256) { /* recommended */
        k += 16;
        constants = sigma;
    } else { /* kbits == 128 */
        constants = tau;
    }
    x->input[8] = U8TO32_LITTLE(k + 0);
    x->input[9] = U8TO32_LITTLE(k + 4);
    x->input[10] = U8TO32_LITTLE(k + 8);
    x->input[11] = U8TO32_LITTLE(k + 12);
    x->input[0] = U8TO32_LITTLE(constants + 0);
    x->input[1] = U8TO32_LITTLE(constants + 4);
    x->input[2] = U8TO32_LITTLE(constants + 8);
    x->input[3] = U8TO32_LITTLE(constants + 12);
    if (iv) {
        x->input[14] = U8TO32_LITTLE(iv + 0);
        x->input[15] = U8TO32_LITTLE(iv + 4);
    } else {
        x->input[14] = 0;
        x->input[15] = 0;
    }
}

void chacha8_get_keystream(const struct chacha8_ctx *x, uint64_t pos, uint32_t n_blocks, uint8_t *c)
{
    // std::cout << "chacha8_get_keystream " << std::endl;
    uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
    uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
    int i;

    j0 = x->input[0];
    j1 = x->input[1];
    j2 = x->input[2];
    j3 = x->input[3];
    j4 = x->input[4];
    j5 = x->input[5];
    j6 = x->input[6];
    j7 = x->input[7];
    j8 = x->input[8];
    j9 = x->input[9];
    j10 = x->input[10];
    j11 = x->input[11];
    j12 = pos;
    j13 = pos >> 32;
    j14 = x->input[14];
    j15 = x->input[15];

    while (n_blocks--) {
        x0 = j0;
        x1 = j1;
        x2 = j2;
        x3 = j3;
        x4 = j4;
        x5 = j5;
        x6 = j6;
        x7 = j7;
        x8 = j8;
        x9 = j9;
        x10 = j10;
        x11 = j11;
        x12 = j12;
        x13 = j13;
        x14 = j14;
        x15 = j15;
        for (i = 8; i > 0; i -= 2) {
            QUARTERROUND(x0, x4, x8, x12);
            QUARTERROUND(x1, x5, x9, x13);
            QUARTERROUND(x2, x6, x10, x14);
            QUARTERROUND(x3, x7, x11, x15);
            QUARTERROUND(x0, x5, x10, x15);
            QUARTERROUND(x1, x6, x11, x12);
            QUARTERROUND(x2, x7, x8, x13);
            QUARTERROUND(x3, x4, x9, x14);
        }
        x0 = PLUS(x0, j0);
        x1 = PLUS(x1, j1);
        x2 = PLUS(x2, j2);
        x3 = PLUS(x3, j3);
        x4 = PLUS(x4, j4);
        x5 = PLUS(x5, j5);
        x6 = PLUS(x6, j6);
        x7 = PLUS(x7, j7);
        x8 = PLUS(x8, j8);
        x9 = PLUS(x9, j9);
        x10 = PLUS(x10, j10);
        x11 = PLUS(x11, j11);
        x12 = PLUS(x12, j12);
        x13 = PLUS(x13, j13);
        x14 = PLUS(x14, j14);
        x15 = PLUS(x15, j15);

        j12 = PLUSONE(j12);
        if (!j12) {
            j13 = PLUSONE(j13);
            /* stopping at 2^70 bytes per nonce is user's responsibility */
        }

        U32TO8_LITTLE(c + 0, x0);
        U32TO8_LITTLE(c + 4, x1);
        U32TO8_LITTLE(c + 8, x2);
        U32TO8_LITTLE(c + 12, x3);
        U32TO8_LITTLE(c + 16, x4);
        U32TO8_LITTLE(c + 20, x5);
        U32TO8_LITTLE(c + 24, x6);
        U32TO8_LITTLE(c + 28, x7);
        U32TO8_LITTLE(c + 32, x8);
        U32TO8_LITTLE(c + 36, x9);
        U32TO8_LITTLE(c + 40, x10);
        U32TO8_LITTLE(c + 44, x11);
        U32TO8_LITTLE(c + 48, x12);
        U32TO8_LITTLE(c + 52, x13);
        U32TO8_LITTLE(c + 56, x14);
        U32TO8_LITTLE(c + 60, x15);

        c += 64;
    }
}


