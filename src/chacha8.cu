#include "hip/hip_runtime.h"
// CUDA libraries.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Include associated header file.
#include "../include/chacha8.cuh"

#define MAX_ARRAY_SIZE 256

// There is a multiply relation between the size of c (output) and n_blocks
// n_blocks = 1, sizeof(c) = uint8_t * 64 * 1
// n_blocks = 2, sizeof(c) = uint8_t * 64 * 2
// n_blocks = 3, sizeof(c) = uint8_t * 64 * 3
#define SIZE_OF_OUTPUT_PER_BLOCK 64

#define U32TO32_LITTLE(v) (v)
#define U8TO32_LITTLE(p) (*(const uint32_t *)(p))
#define U32TO8_LITTLE(p, v) (((uint32_t *)(p))[0] = U32TO32_LITTLE(v))
#define ROTL32(v, n) (((v) << (n)) | ((v) >> (32 - (n))))

#define ROTATE(v, c) (ROTL32(v, c))
#define XOR(v, w) ((v) ^ (w))
#define PLUS(v, w) ((v) + (w))
#define PLUSONE(v) (PLUS((v), 1))

#define QUARTERROUND(a, b, c, d) \
    a = PLUS(a, b);              \
    d = ROTATE(XOR(d, a), 16);   \
    c = PLUS(c, d);              \
    b = ROTATE(XOR(b, c), 12);   \
    a = PLUS(a, b);              \
    d = ROTATE(XOR(d, a), 8);    \
    c = PLUS(c, d);              \
    b = ROTATE(XOR(b, c), 7)

// This is the GPU device code
__global__ void Kernel_Print(int * block_dim, int * thread_id, int * grid_dim)
{
    printf("blockDim x:%d, y:%d, z:%d\n", blockDim.x, blockDim.y, blockDim.z);
    printf("threadIdx x:%d, y:%d, z:%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("gridDim x:%d, y:%d, z:%d\n", gridDim.x, gridDim.y, gridDim.z);

    block_dim[0] = blockDim.x;
    block_dim[1] = blockDim.y;
    block_dim[2] = blockDim.z;

    thread_id[0] = threadIdx.x;
    thread_id[1] = threadIdx.y;
    thread_id[2] = threadIdx.z;

    grid_dim[0] = gridDim.x;
    grid_dim[1] = gridDim.y;
    grid_dim[2] = gridDim.z;
}


// This is the GPU device code
__global__ void chacha8_get_keystream_cuda( struct chacha8_ctx *x, uint64_t *pos, uint32_t *n_blocks, uint8_t **c)
{
    int idx = threadIdx.x;
    printf("[chacha8_get_keystream_cuda] i = %d\n", idx);

    uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
    uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
    int i;

    j0 = x[idx].input[0];
    j1 = x[idx].input[1];
    j2 = x[idx].input[2];
    j3 = x[idx].input[3];
    j4 = x[idx].input[4];
    j5 = x[idx].input[5];
    j6 = x[idx].input[6];
    j7 = x[idx].input[7];
    j8 = x[idx].input[8];
    j9 = x[idx].input[9];
    j10 = x[idx].input[10];
    j11 = x[idx].input[11];
    j12 = pos[idx];
    j13 = pos[idx] >> 32;
    j14 = x[idx].input[14];
    j15 = x[idx].input[15];

    while (n_blocks[idx]--) {
        x0 = j0;
        x1 = j1;
        x2 = j2;
        x3 = j3;
        x4 = j4;
        x5 = j5;
        x6 = j6;
        x7 = j7;
        x8 = j8;
        x9 = j9;
        x10 = j10;
        x11 = j11;
        x12 = j12;
        x13 = j13;
        x14 = j14;
        x15 = j15;
        for (i = 8; i > 0; i -= 2) {
            QUARTERROUND(x0, x4, x8, x12);
            QUARTERROUND(x1, x5, x9, x13);
            QUARTERROUND(x2, x6, x10, x14);
            QUARTERROUND(x3, x7, x11, x15);
            QUARTERROUND(x0, x5, x10, x15);
            QUARTERROUND(x1, x6, x11, x12);
            QUARTERROUND(x2, x7, x8, x13);
            QUARTERROUND(x3, x4, x9, x14);
        }
        x0 = PLUS(x0, j0);
        x1 = PLUS(x1, j1);
        x2 = PLUS(x2, j2);
        x3 = PLUS(x3, j3);
        x4 = PLUS(x4, j4);
        x5 = PLUS(x5, j5);
        x6 = PLUS(x6, j6);
        x7 = PLUS(x7, j7);
        x8 = PLUS(x8, j8);
        x9 = PLUS(x9, j9);
        x10 = PLUS(x10, j10);
        x11 = PLUS(x11, j11);
        x12 = PLUS(x12, j12);
        x13 = PLUS(x13, j13);
        x14 = PLUS(x14, j14);
        x15 = PLUS(x15, j15);

        j12 = PLUSONE(j12);
        if (!j12) {
            j13 = PLUSONE(j13);
            /* stopping at 2^70 bytes per nonce is user's responsibility */
        }

        U32TO8_LITTLE(c[idx] + 0, x0);
        U32TO8_LITTLE(c[idx] + 4, x1);
        U32TO8_LITTLE(c[idx] + 8, x2);
        U32TO8_LITTLE(c[idx] + 12, x3);
        U32TO8_LITTLE(c[idx] + 16, x4);
        U32TO8_LITTLE(c[idx] + 20, x5);
        U32TO8_LITTLE(c[idx] + 24, x6);
        U32TO8_LITTLE(c[idx] + 28, x7);
        U32TO8_LITTLE(c[idx] + 32, x8);
        U32TO8_LITTLE(c[idx] + 36, x9);
        U32TO8_LITTLE(c[idx] + 40, x10);
        U32TO8_LITTLE(c[idx] + 44, x11);
        U32TO8_LITTLE(c[idx] + 48, x12);
        U32TO8_LITTLE(c[idx] + 52, x13);
        U32TO8_LITTLE(c[idx] + 56, x14);
        U32TO8_LITTLE(c[idx] + 60, x15);

        c[idx] += 64;
    }
}



void get_chacha8_key(struct chacha8_ctx *_x, uint64_t *_pos, uint32_t *_n_blocks, uint8_t **_c, int array_size)
{
    // 
    std::cout << "Size of uint64_t:" << sizeof(uint64_t) << std::endl;
    std::cout << "Size of * uint64_t:" << sizeof(_pos) << std::endl;
    std::cout << "Size of uint32_t:" << sizeof(uint32_t) << std::endl;
    std::cout << "Size of * uint32_t:" << sizeof(_n_blocks) << std::endl;
    std::cout << "Size of struct chacha8_ctx:" << sizeof(struct chacha8_ctx) << std::endl;

    if (array_size > MAX_ARRAY_SIZE)
    {
        std::cout << "Array size out of bound" << std::endl;
        return;
    }

    struct chacha8_ctx *x = _x;
    uint64_t *pos = _pos;
    uint32_t *n_blocks = _n_blocks;
    uint8_t **c = _c;
    int thread_block = array_size;

    // Has to handle error if memory allocation failed
    hipError_t error;

    error = hipMalloc((void**) &pos, array_size * sizeof(uint64_t));
    if (error)
    {
        std::cout << "hipMalloc fail at pos error: " << error << std::endl; 
        return;
    }

    error = hipMalloc((void**) &n_blocks, array_size * sizeof(uint32_t));
    if (error)
    {
        std::cout << "hipMalloc fail at n_blocks error: " << error << std::endl; 
        return;
    }

    error = hipMalloc((void**) &x, array_size * sizeof(struct chacha8_ctx));
    if (error)
    {
        std::cout << "hipMalloc fail at x error: " << error << std::endl; 
        return;
    }
    std::cout << "happen error: " << std::endl; 
    error = hipMalloc((void**) &c, SIZE_OF_OUTPUT_PER_BLOCK * _n_blocks[0]);
    if (error)
    {
        std::cout << "hipMalloc fail at c error: " << error << std::endl; 
        return;
    }
    std::cout << "happen after error: " << std::endl; 
    error = hipMemcpy(pos, _pos, array_size * sizeof(uint64_t), hipMemcpyHostToDevice);
    if (error)
    {
        std::cout << "hipMemcpy fail at pos error: " << error << std::endl; 
        return;
    }

    error = hipMemcpy(n_blocks, _n_blocks, array_size * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (error)
    {
        std::cout << "hipMemcpy fail at n_blocks error: " << error << std::endl; 
        return;
    }

    error = hipMemcpy(x, _x, array_size * sizeof(struct chacha8_ctx), hipMemcpyHostToDevice);
    if (error)
    {
        std::cout << "hipMemcpy fail at x error: " << error << std::endl; 
        return;
    }



    // Calculate blocksize and gridsize.
    // dim3 blockSize(512, 1, 1);
    // dim3 gridSize(512 / array_size + 1, 1);

    chacha8_get_keystream_cuda<<<1, thread_block>>>(x, pos, n_blocks, c);

    // Copy result to output
    error = hipMemcpy(_c[0], c[0], SIZE_OF_OUTPUT_PER_BLOCK * n_blocks[0], hipMemcpyDeviceToHost);
    if (error)
    {
        std::cout << "hipMemcpy fail at x error: " << error << std::endl; 
        return;
    }
    int block_dim[3] = {1, 2, 3};
    int thread_id[3] = {1, 2, 3};
    int grid_dim[3] = {1, 2, 3};


    Kernel_Print<<<1, thread_block>>>(block_dim, thread_id, grid_dim);
    std::cout << "block_dim.x: " << block_dim[0] << ", block_dim.y: " << block_dim[1] << ", block_dim.z: " << block_dim[2] << std::endl;
    std::cout << "thread_id.x: " << thread_id[0] << ", thread_id.y: " << thread_id[1] << ", thread_id.z: " << thread_id[2] << std::endl;
    std::cout << "grid_dim.x: " << grid_dim[0] << ", grid_dim.y: " << grid_dim[1] << ", grid_dim.z: " << grid_dim[2] << std::endl;

    // free memory
}

// This is host code
void chacha8_keysetup(struct chacha8_ctx *x, const uint8_t *k, uint32_t kbits, const uint8_t *iv)
{
    const char *constants;
    static const char sigma[17] = "expand 32-byte k";
    static const char tau[17] = "expand 16-byte k";

    x->input[4] = U8TO32_LITTLE(k + 0);
    x->input[5] = U8TO32_LITTLE(k + 4);
    x->input[6] = U8TO32_LITTLE(k + 8);
    x->input[7] = U8TO32_LITTLE(k + 12);
    if (kbits == 256) { /* recommended */
        k += 16;
        constants = sigma;
    } else { /* kbits == 128 */
        constants = tau;
    }
    x->input[8] = U8TO32_LITTLE(k + 0);
    x->input[9] = U8TO32_LITTLE(k + 4);
    x->input[10] = U8TO32_LITTLE(k + 8);
    x->input[11] = U8TO32_LITTLE(k + 12);
    x->input[0] = U8TO32_LITTLE(constants + 0);
    x->input[1] = U8TO32_LITTLE(constants + 4);
    x->input[2] = U8TO32_LITTLE(constants + 8);
    x->input[3] = U8TO32_LITTLE(constants + 12);
    if (iv) {
        x->input[14] = U8TO32_LITTLE(iv + 0);
        x->input[15] = U8TO32_LITTLE(iv + 4);
    } else {
        x->input[14] = 0;
        x->input[15] = 0;
    }
}


